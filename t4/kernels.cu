#include <hip/hip_runtime.h>

#include "stdio.h"

#define WARP_SIZE 32

#define _CSR_VEC_ALT

#define TILE_SIZE 512

extern "C" void CSR_matvec(int N, int nnz, int* start, int* indices, float* data, float* x, float *y, int bVectorized);
extern "C" void CSR_create(int N, int nnz, int* start, int * indices, float * data , float * x , float * y, int** start_d, int **indices_d, float **data_d, float **x_d, float **y_d);
extern "C" void CSR_kernel(int N, int nnz, int* start_d, int * indices_d, float * data_d , float * x_d , float * y_d, int bVectorized);
extern "C" void CSR_destroy(int* start_d, int* indices_d, float* data_d, float* x_d, float* y_d);

extern "C" void ELL_matvec(int N, int num_cols_per_row , int * indices, float * data , float * x , float * y);
extern "C" void ELL_create(int N, int num_cols_per_row, int * indices, float * data , float * x , float * y, int **indices_d, float **data_d, float **x_d, float **y_d);
extern "C" void ELL_kernel(int N, int num_cols_per_row , int * indices_d, float * data_d , float * x_d , float * y_d);
extern "C" void ELL_destroy(int* indices_d, float* data_d, float* x_d, float* y_d);

/**
 * Custom CUDA error check wrapper.
 */
#define checkCUDAError() do {                           \
 hipError_t error = hipGetLastError();               \
 if (error != hipSuccess) {                            \
   printf("(CUDA) %s", hipGetErrorString(error)); \
   printf(" (" __FILE__ ":%d)\n", __LINE__);  \
  }\
} while (0)

/**
 * Cuda kernel for: CSR_s(A)x = y
 */
__global__ void k_csr_mat_vec_mm(const int N, int *start, int* indices, float *data, float *x, float* y) {
	int row = blockDim.x * blockIdx.x + threadIdx.x ;

	if ( row < N ){
		float dot = 0;
		int row_start = start [ row ];
		int row_end = start [ row+1];

		for ( int jj = row_start ; jj < row_end ; jj ++) {
			dot += data [ jj ] * x [ indices [ jj ]];
		}

		y[row] = dot ;
	}
}

/**
 * Cuda kernel for: CSR_v(A)x = y
 */

#if !defined(_CSR_VEC_ALT)
    __global__ void k_csr2_mat_vec_mm(const int N, int *start, int* indices, float *data, float *x, float* y) {
      __shared__ float vals[TILE_SIZE];
       
      int thread_id = TILE_SIZE * blockIdx.x + threadIdx.x;
      int warp_id = thread_id / WARP_SIZE;
      int lane = thread_id & (WARP_SIZE - 1);
      int row = warp_id;

      if (row < N) {
        int row_start = start[row];
        int row_end = start[row + 1];

	    // compute running sum per thread
	    vals[threadIdx.x] = 0;

	    for (int jj = row_start + lane; jj < row_end; jj += WARP_SIZE) {
	      vals[threadIdx.x] += data[jj] * x[indices[jj]];
	    }

        // parallel reduction in shared memory
        for (int d = WARP_SIZE >> 1; d >= 1; d >>= 1) {
          if (lane < d) vals[threadIdx.x] += vals[threadIdx.x + d];
        }

        // first thread in a warp writes the result
        if (lane == 0) {
          y[row] = vals[threadIdx.x];
        }
      }
    }
#else
    /**
     * Cuda kernel for: CSR_v(A)x = y
     */
    __global__ void k_csr2a_mat_vec_mm(const int N, int *start, int* indices, float *data, float *x, float* y) {
      __shared__ float vals[TILE_SIZE / WARP_SIZE][WARP_SIZE];
       
      int warp_id = TILE_SIZE / WARP_SIZE * blockIdx.x + threadIdx.y;
      int lane = threadIdx.x;
      int row = warp_id;

      if (row < N) {
        int row_start = start[row];
        int row_end = start[row + 1];

	    // compute running sum per thread
	    vals[threadIdx.y][lane] = 0;

	    for (int jj = row_start + lane; jj < row_end; jj += WARP_SIZE) {
	      vals[threadIdx.y][lane] += data[jj] * x[indices[jj]];
	    }

        // parallel reduction in shared memory
        for (int d = WARP_SIZE >> 1; d >= 1; d >>= 1) {
          if (lane < d) vals[threadIdx.y][lane] += vals[threadIdx.y][lane + d];
        }

        // first thread in a warp writes the result
        if (lane == 0) {
          y[row] = vals[threadIdx.y][0];
        }
      }
    }
#endif

/**
 * Cuda kernel for: ELL(A)x = y
 */
__global__ void k_ell_mat_vec_mm (const int N, const int num_cols_per_row, int * indices, float * data, float * x, float * y) {
	int row = blockDim.x * blockIdx.x + threadIdx.x;

	if (row < N) {
		float dot = 0;
		for (int n = 0; n < num_cols_per_row ; n++) {
			// TODO: Uncomment and complete the sample.
			int col = indices[row * num_cols_per_row + n];
			float val = data[row * num_cols_per_row + n];
			if (val != 0) {
				dot += val * x[row];
			}
		}
		y[row] = dot;
	}
}

/**
 * Perform: CSR(A)x = y
 */
void CSR_matvec(const int N, const int nnz, int* start, int * indices, float * data , float * x , float * y, const int bVectorized) {
	int *start_d, *indices_d;
	float *data_d, *x_d, *y_d;

	CSR_create(N, nnz, start, indices, data, x, y, &start_d, &indices_d, &data_d, &x_d, &y_d);

	CSR_kernel(N, nnz, start_d, indices_d, data_d, x_d, y_d, bVectorized);

	hipMemcpy(y, y_d, N * sizeof(float), hipMemcpyDeviceToHost);
	checkCUDAError();

	CSR_destroy(start_d, indices_d, data_d, x_d, y_d);
}


/**
 * Create CSR matrix
 */
void CSR_create(const int N, const int nnz,
		int * start, int * indices, float * data , float * x , float * y, 
		int ** start_d, int ** indices_d, float **data_d, float **x_d, float **y_d) {

	/************************/
	/* copy to device       */
	/************************/

	hipMalloc((void **) start_d, (N+1) * sizeof(int));
	checkCUDAError();
	hipMemcpy(*start_d, start, (N+1) * sizeof(int), hipMemcpyHostToDevice);
	checkCUDAError();

	hipMalloc((void **) indices_d, nnz * sizeof(int));
	checkCUDAError();
	hipMemcpy(*indices_d, indices, nnz * sizeof(int), hipMemcpyHostToDevice);
	checkCUDAError();

	hipMalloc((void **) data_d, nnz * sizeof(float));
	checkCUDAError();
	hipMemcpy(*data_d, data, nnz * sizeof(float), hipMemcpyHostToDevice);
	checkCUDAError();

	hipMalloc((void **) x_d, N * sizeof(float));
	checkCUDAError();
	hipMemcpy(*x_d, x, N * sizeof(float), hipMemcpyHostToDevice);
	checkCUDAError();

	hipMalloc((void **) y_d, N * sizeof(float));
	checkCUDAError();
	hipMemcpy(*y_d, y, N * sizeof(float) , hipMemcpyHostToDevice);
	checkCUDAError();
}

/**
 * Perform: CSR(A)x = y
 */
void CSR_kernel(const int N, const int nnz, int * start_d , int * indices_d, float * data_d , float * x_d , float * y_d, const int bVectorized) {
	if (bVectorized) {
        //#threads = #rows * #threads per row (= N * WARP_SIZE)
		dim3 grid((N * WARP_SIZE - 1)/TILE_SIZE + 1, 1, 1);

#       if !defined(_CSR_VEC_ALT)
		    dim3 block(TILE_SIZE, 1, 1);

		    k_csr2_mat_vec_mm <<< grid, block >>> (N, start_d, indices_d, data_d, x_d, y_d);
#       else
		    dim3 block(WARP_SIZE, TILE_SIZE / WARP_SIZE, 1);

		    k_csr2a_mat_vec_mm <<< grid, block >>> (N, start_d, indices_d, data_d, x_d, y_d);
#       endif
	} else {
		//#threads = #rows (= N)
		dim3 grid((N - 1)/TILE_SIZE + 1, 1, 1);
		dim3 block(TILE_SIZE, 1, 1);

		k_csr_mat_vec_mm <<< grid, block >>> (N, start_d, indices_d, data_d, x_d, y_d);
	}

	checkCUDAError();
}

/**
 * Destroy CSR matrix
 */
void CSR_destroy(int* start_d, int* indices_d, float* data_d, float* x_d, float* y_d) {
	hipFree(start_d);
	hipFree(indices_d);
	hipFree(data_d);
	hipFree(x_d);
	hipFree(y_d);
}

/**
 * Perform: ELL(A)x = y
 */
void ELL_matvec(const int N, const int num_cols_per_row , int * indices, float * data , float * x , float * y) {
	int *indices_d;
	float *data_d, *x_d, *y_d;

	ELL_create(N, num_cols_per_row, indices, data, x, y, &indices_d, &data_d, &x_d, &y_d);

	ELL_kernel(N, num_cols_per_row, indices_d, data_d, x_d, y_d);

	hipMemcpy(y, y_d, N * sizeof(float), hipMemcpyDeviceToHost);
	checkCUDAError();

	ELL_destroy(indices_d, data_d, x_d, y_d);
}


/**
 * Create ELL matrix
 */
void ELL_create(const int N, const int num_cols_per_row,
		int * indices, float * data , float * x , float * y, 
		int ** indices_d, float **data_d, float **x_d, float **y_d) {

	hipMalloc((void **) indices_d, N * num_cols_per_row * sizeof(int));
	checkCUDAError();
	hipMemcpy(*indices_d, indices, N * num_cols_per_row * sizeof(int), hipMemcpyHostToDevice);
	checkCUDAError();

	hipMalloc((void **) data_d, N * num_cols_per_row * sizeof(float));
	checkCUDAError();
	hipMemcpy(*data_d, data, N * num_cols_per_row * sizeof(float), hipMemcpyHostToDevice);
	checkCUDAError();

	hipMalloc((void **) x_d, N * sizeof(float));
	checkCUDAError();
	hipMemcpy(*x_d, x, N * sizeof(float), hipMemcpyHostToDevice);
	checkCUDAError();

	hipMalloc((void **) y_d, N * sizeof(float));
	checkCUDAError();
	hipMemcpy(*y_d, y, N * sizeof(float), hipMemcpyHostToDevice);
	checkCUDAError();
}

/**
 * Perform: ELL(A)x = y
 */
void ELL_kernel(int N, int num_cols_per_row , int * indices_d, float * data_d , float * x_d , float * y_d) {
	//round grid size N/TILE_SIZE up
	dim3 grid((N - 1)/TILE_SIZE + 1, 1, 1);
	dim3 block(TILE_SIZE, 1, 1);

	k_ell_mat_vec_mm <<< grid, block >>> (N, num_cols_per_row, indices_d, data_d , x_d, y_d);
	checkCUDAError();
}

/**
 * Destroy ELL matrix
 */
void ELL_destroy(int* indices_d, float* data_d, float* x_d, float* y_d) {
	hipFree(indices_d);
	hipFree(data_d);
	hipFree(x_d);
	hipFree(y_d);
}

